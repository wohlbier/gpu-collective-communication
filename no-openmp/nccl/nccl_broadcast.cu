#include <stdio.h>
#include <mpi.h>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                                 \
    hipError_t e = cmd;                                    \
    if( e != hipSuccess ) {                                \
        printf("Failed: Cuda error %s:%d '%s'\n",           \
                __FILE__,__LINE__,hipGetErrorString(e));   \
        exit(EXIT_FAILURE);                                 \
    }                                                       \
} while(0)


#define NCCLCHECK(cmd) do {                                 \
    ncclResult_t r = cmd;                                   \
    if (r!= ncclSuccess) {                                  \
        printf("Failed, NCCL error %s:%d '%s'\n",           \
                __FILE__,__LINE__,ncclGetErrorString(r));   \
        exit(EXIT_FAILURE);                                 \
    }                                                       \
} while(0)


int main(int argc, char* argv[])
{
    if (argc < 2) {
        fprintf(stderr, "need a value for N\n");
        exit(EXIT_FAILURE);
    }
    const size_t size = atoi(argv[1]);

    int nDev = 0;
    if (argc > 2) {
        nDev = atoi(argv[2]);
    }
    int deviceCount;
    CUDACHECK( hipGetDeviceCount(&deviceCount) );
    if (deviceCount < nDev || nDev == 0) {
        fprintf(stderr, "changing nDev from %d to %d\n", nDev, deviceCount);
        nDev = deviceCount;
    }


    // init MPI
    int mpi_rank, mpi_size;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);


    ncclUniqueId id;
    if (mpi_rank == 0) ncclGetUniqueId(&id);
    MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);


    int *device_counts = (int*)malloc(mpi_size * sizeof(int));
    MPI_Allgather(&nDev, 1, MPI_INT,
            device_counts, 1, MPI_INT, MPI_COMM_WORLD);
    int device_rank_start = 0, total_device_count = 0;
    for (int i = 0; i < mpi_size; ++i) {
        if (i < mpi_rank) device_rank_start += device_counts[i];
        total_device_count += device_counts[i];
    }


    //initializing NCCL, group API is required around ncclCommInitRank as it is
    //called across multiple GPUs in each thread/process
    ncclComm_t *comms = (ncclComm_t*)malloc(nDev * sizeof(ncclComm_t));
    NCCLCHECK(ncclGroupStart());
    for (int i=0; i<nDev; i++) {
        CUDACHECK(hipSetDevice(i));
        NCCLCHECK(ncclCommInitRank(comms+i, total_device_count, id, device_rank_start++));
    }
    NCCLCHECK(ncclGroupEnd());


    //allocating and initializing device buffers
    float** sendbuff = (float**)malloc(nDev * sizeof(float*));
    float** recvbuff = (float**)malloc(nDev * sizeof(float*));
    hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);
    hipEvent_t* start = (hipEvent_t*)malloc(sizeof(hipEvent_t)*nDev);
    hipEvent_t* stop = (hipEvent_t*)malloc(sizeof(hipEvent_t)*nDev);


    for (int i = 0; i < nDev; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
        CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
        CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
        CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
        CUDACHECK(hipStreamCreate(s+i));
        CUDACHECK(hipEventCreate(start+i));
        CUDACHECK(hipEventCreate(stop+i));
    }


    // start timers
    for (int i = 0; i < nDev; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipEventRecord(start[i], s[i]));
    }


    //calling NCCL communication API. Group API is required when using
    //multiple devices per thread
    NCCLCHECK(ncclGroupStart());
    for (int i = 0; i < nDev; ++i)
        NCCLCHECK(ncclBroadcast((const void*)sendbuff[i], (void*)recvbuff[i],
                    size, ncclFloat, 0, comms[i], s[i]));
    NCCLCHECK(ncclGroupEnd());


    //synchronizing on CUDA streams to wait for completion of NCCL operation
    for (int i = 0; i < nDev; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipEventRecord(stop[i], s[i]));
        CUDACHECK(hipStreamSynchronize(s[i]));
    }


    //free device buffers
    for (int i = 0; i < nDev; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipFree(sendbuff[i]));
        CUDACHECK(hipFree(recvbuff[i]));
    }
    free(sendbuff);
    free(recvbuff);


    //finalizing NCCL
    for(int i = 0; i < nDev; ++i)
        ncclCommDestroy(comms[i]);


    // report timings
    for (int i = 0; i < nDev; ++i) {
        float time;
        CUDACHECK(hipEventElapsedTime(&time, start[i], stop[i]));
        printf("dev %d: %f ms\n", i, time);
    }


    free(comms);


    MPI_Finalize();


    printf("[MPI Rank %d] Success \n", mpi_rank);
    return 0;
}
