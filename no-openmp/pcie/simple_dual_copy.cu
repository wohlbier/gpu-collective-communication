#include <hip/hip_runtime.h>
#include <iostream>

#include "common.h"

int main(int argc, char *argv[]) {
    const size_t count = 10000000;
    const size_t bufSize = count * sizeof(tx_type);

    hipStream_t s[2];
    hipEvent_t start[2];
    hipEvent_t stop[2];

    tx_type *d_buffers[2];

    for (int d = 0; d < 2; ++d) {
        CUDACHECK(hipSetDevice(d));
        CUDACHECK(hipStreamCreateWithFlags(&s[d], hipStreamNonBlocking));
        CUDACHECK(hipEventCreate(&start[d]));
        CUDACHECK(hipEventCreate(&stop[d]));
        CUDACHECK(hipMalloc(&d_buffers[d], bufSize));
    }

    tx_type *h_original, *h_result;
    CUDACHECK(hipHostMalloc(&h_original, bufSize));
    CUDACHECK(hipHostMalloc(&h_result, bufSize));

    for (size_t i = 0; i < count; ++i) {
        h_original[i] = i;
    }
    CUDACHECK(hipMemcpy(d_buffers[0], h_original, bufSize, hipMemcpyDefault));

    const size_t half = count / 2;
    const size_t halfBufSize = half * sizeof(tx_type);

    for (int d = 0; d < 2; ++d) {
        CUDACHECK(hipSetDevice(d));
        CUDACHECK(hipEventRecord(start[d], s[d]));
    }

    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipMemcpyAsync(d_buffers[1], d_buffers[0],
                halfBufSize, hipMemcpyDefault, s[0]));
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMemcpyAsync(d_buffers[1]+half, d_buffers[0]+half,
                (bufSize - halfBufSize), hipMemcpyDefault, s[1]));
    //CUDACHECK(hipSetDevice(0));
    //CUDACHECK(hipMemcpyPeerAsync(d_buffers[1], 1, d_buffers[0], 0,
    //            halfBufSize, s[0]));
    //CUDACHECK(hipSetDevice(1));
    //CUDACHECK(hipMemcpyPeerAsync(d_buffers[1]+half, 1, d_buffers[0]+half, 0,
    //            (bufSize - halfBufSize), s[1]));

    for (int d = 0; d < 2; ++d) {
        CUDACHECK(hipSetDevice(d));
        CUDACHECK(hipEventRecord(stop[d], s[d]));
    }

    for (int d = 0; d < 2; ++d) {
        CUDACHECK(hipSetDevice(d));
        CUDACHECK(hipDeviceSynchronize());
        float ms;
        CUDACHECK(hipEventElapsedTime(&ms, start[d], stop[d]));
        std::cout << ms << " ms" << std::endl;
    }

    CUDACHECK(hipMemcpy(h_result, d_buffers[1], bufSize, hipMemcpyDefault));
    for (size_t i = 0; i < count; ++i) {
        if (h_original[i] != h_result[i]) {
            std::cerr << "data mismatch at idx=" << i << std::endl;
            break;
        }
    }

    for (int d = 0; d < 2; ++d) {
        CUDACHECK(hipFree(d_buffers[d]));
    }
    CUDACHECK(hipHostFree(h_result));
    CUDACHECK(hipHostFree(h_original));

    return 0;
}
