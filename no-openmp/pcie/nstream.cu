#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#include "common.h"

int main(int argc, char *argv[]) {
    std::cout << "starting nstream" << std::endl;

    const int nDevs = 2;
    const int N = atoi(argv[1]);
    const int nStreams = atoi(argv[2]);

    // make sure we have enough devices
    int sysDevices;
    CUDACHECK(hipGetDeviceCount(&sysDevices));

    if (sysDevices < nDevs) {
        std::cerr << "need at least 2 cuda devices" << std::endl;
        exit(EXIT_FAILURE);
    }

    // create space for things
    const size_t numBytes = N * sizeof(tx_type);
    const size_t chunkCount = N / nStreams;
    tx_type *h_data;
    CUDACHECK(hipHostMalloc(&h_data, numBytes));

    tx_type *d_data[nDevs];
    hipStream_t s[nDevs][nStreams];
    hipEvent_t start[nDevs][nStreams];
    hipEvent_t stop[nDevs][nStreams];

    for (int i = 0; i < nDevs; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipMalloc(&d_data[i], numBytes));
        for (int j = 0; j < nStreams; ++j) {
            CUDACHECK(hipStreamCreate(&s[i][j]));
            CUDACHECK(hipEventCreate(&start[i][j]));
            CUDACHECK(hipEventCreate(&stop[i][j]));
        }
    }

    // set host data to be something to check delivery
    CUDACHECK(hipMemset(h_data, 1, numBytes));
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipMemcpy(d_data[0], h_data, numBytes, hipMemcpyDefault));

    // do the transfers
    for (int i = 1; i < nDevs; ++i) {
        CUDACHECK(hipSetDevice(i));
        for (int j = 0; j < nStreams; ++j) {
            CUDACHECK(hipEventRecord(start[i][j], s[i][j]));
            CUDACHECK(hipMemcpyAsync(d_data[i]+j*chunkCount, d_data[0]+j*chunkCount, chunkCount * sizeof(tx_type), hipMemcpyDefault, s[i][j]));
            CUDACHECK(hipEventRecord(stop[i][j], s[i][j]));
        }
    }

    // print timing info
    for (int dev = 1; dev < nDevs; ++dev) {
        CUDACHECK(hipSetDevice(dev));
        CUDACHECK(hipDeviceSynchronize());
        for (int i = 0; i < nStreams; ++i) {
            float ms;
            CUDACHECK(hipEventElapsedTime(&ms, start[dev][i], stop[dev][i]));
            printf("time for dev %d stream %d: %f\n", dev, i, ms);
        }
    }

    for (int i=0; i<nDevs; ++i) CUDACHECK(hipFree(d_data[i]));

    return 0;
}
