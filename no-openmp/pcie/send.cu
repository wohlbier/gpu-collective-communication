#include <hip/hip_runtime.h>
#include <iostream>

#include "common.h"

/**
 * Gear things up as a send - see if we can actually do better with having
 * GPUs in a push/pull config instead of just pull
 */

int main(int argc, char *argv[]) {
    const int nDevs = 2;

    // make sure we have enough devices
    int numDevices;
    CUDACHECK(hipGetDeviceCount(&numDevices));

    if (numDevices < nDevs) {
        std::cout << "not enough devices" << std::endl;
        exit(EXIT_FAILURE);
    }

    // some arg parsing
    const size_t N = atoi(argv[1]);
    int nStreams = 1;
    if (argc > 2)
        nStreams = atoi(argv[2]);

    // divide among devs and then streams
    int devChunkSize = N / nDevs;
    int streamChunkSize = devChunkSize / nStreams;

    tx_type *h_data;
    CUDACHECK(hipHostMalloc(&h_data, N * sizeof(tx_type)));
    CUDACHECK(hipMemset(h_data, 1., N * sizeof(tx_type)));

    tx_type *d_data[nDevs];
    hipEvent_t *start[nDevs];
    hipEvent_t *stop[nDevs];
    hipStream_t **s = (hipStream_t**)malloc(nDevs * sizeof(hipStream_t*));
    for (int device = 0; device < nDevs; ++device) {
        CUDACHECK(hipSetDevice(device));
        CUDACHECK(hipMalloc(&d_data[device], N * sizeof(tx_type)));

        s[device] = (hipStream_t*)malloc(nStreams * sizeof(hipStream_t));
        start[device] = (hipEvent_t*)malloc(nStreams * sizeof(hipStream_t));
        stop[device] = (hipEvent_t*)malloc(nStreams * sizeof(hipStream_t));
        for (int i = 0; i < nStreams; ++i) {
            CUDACHECK(hipStreamCreate(s[device]+i));
            CUDACHECK(hipEventCreate(start[device]+i));
            CUDACHECK(hipEventCreate(stop[device]+i));
        }
    }

    CUDACHECK(hipMemcpy(d_data[0], h_data, N * sizeof(tx_type), hipMemcpyDefault));


    // do the copy
    // how many ways can I split this up two ways???
    int fDev = 0, fStream = 0, fEvent = 0;
    int sDev = 1, sStream = 0, sEvent = 0;
    CUDACHECK(hipSetDevice(fDev));
    CUDACHECK(hipEventRecord(start[fDev][fEvent], s[fDev][fStream]));
    CUDACHECK(hipSetDevice(sDev));
    CUDACHECK(hipEventRecord(start[sDev][sEvent], s[sDev][sStream]));

    CUDACHECK(hipSetDevice(fDev));
    CUDACHECK(hipMemcpyAsync(d_data[1], d_data[0],
                devChunkSize * sizeof(tx_type), hipMemcpyDefault, s[fDev][fStream]));
    CUDACHECK(hipSetDevice(sDev));
    CUDACHECK(hipMemcpyAsync(d_data[1] + devChunkSize, d_data[0] + devChunkSize,
                (N - devChunkSize) * sizeof(tx_type), hipMemcpyDefault, s[sDev][sStream]));

    CUDACHECK(hipEventRecord(stop[fDev][fEvent], s[fDev][fStream]));
    CUDACHECK(hipEventRecord(stop[sDev][sEvent], s[sDev][sStream]));


    // error checking
    tx_type *h_result;
    CUDACHECK(hipHostMalloc(&h_result, N * sizeof(tx_type)));
    CUDACHECK(hipMemcpy(h_result, d_data[1], N * sizeof(tx_type), hipMemcpyDefault));

    for (size_t i = 0; i < N; ++i) {
        if (h_result[i] != h_data[i]) {
            std::cout << "copy failed" << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    for (int dev = 0; dev < nDevs; ++dev) {
        CUDACHECK(hipSetDevice(dev));
        CUDACHECK(hipDeviceSynchronize());
    }

    float ms;
    CUDACHECK(hipEventElapsedTime(&ms, start[fDev][fEvent], stop[fDev][fEvent]));
    printf("first copy took %f ms\n", ms);

    CUDACHECK(hipEventElapsedTime(&ms, start[sDev][sEvent], stop[sDev][sEvent]));
    printf("second copy took %f ms\n", ms);

    for (int device = 0; device < nDevs; ++device) {
        CUDACHECK(hipFree(d_data[device]));
    }

    CUDACHECK(hipHostFree(h_result));
    CUDACHECK(hipHostFree(h_data));

    for (int dev = 0; dev < nDevs; ++dev) {
        free(s[dev]);
        free(start[dev]);
        free(stop[dev]);
    }
    free(s);

    return 0;
}
