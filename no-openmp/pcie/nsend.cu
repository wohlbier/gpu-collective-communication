/**
 * Testing sending from one to N GPUs in a variety of different ways
 */

#include <hip/hip_runtime.h>
#include <vector>
#include <cstdio>
#include <iostream>
#include <thread>

#include "common.h"

static const size_t defaultCount = 10000000;

template <typename T>
void cudaCopy(int device, hipStream_t stream, T* dest, const T* src, size_t bytes) {
    CUDACHECK(hipSetDevice(device));
    CUDACHECK(hipMemcpyAsync(dest, src, bytes, hipMemcpyDefault, stream));
}

template <typename T>
void cudaCopyRoot(int dev, int nDevs, hipStream_t *streams,
        T** dest, const T* src, size_t bytes)
{
    CUDACHECK(hipSetDevice(dev));
    for (int d = 0; d < nDevs; ++d) {
        if (d != dev)
            CUDACHECK(hipMemcpyAsync(dest[d], src, bytes, hipMemcpyDefault, streams[d]));
    }
}

template <typename T>
void broadcastMST(T **d_buffers, size_t count, int root, int left, int right,
        hipStream_t *stream, hipEvent_t *event)
{
    // exit condition
    if (left == right) return;

    const size_t numBytes = count * sizeof(T);

    // get the midpoint of this segment
    int mid = ((left + right - 1) / 2) + 1;

    if (root < mid) {
        // send to right
        CUDACHECK(hipMemcpyAsync(d_buffers[right], d_buffers[root], numBytes,
                    hipMemcpyDefault, stream[root]));

        // make the next copy wait on this copy to finish
        CUDACHECK(hipEventRecord(event[root], stream[root]));
        CUDACHECK(hipStreamWaitEvent(stream[right], event[root], 0));

        broadcastMST(d_buffers, count, root, left, mid-1, stream, event);
        broadcastMST(d_buffers, count, right, mid, right, stream, event);
    }
    else {
        // send to left
        CUDACHECK(hipMemcpyAsync(d_buffers[left], d_buffers[root], numBytes,
                    hipMemcpyDefault, stream[root]));

        // make the next copy wait on this copy to finish
        CUDACHECK(hipEventRecord(event[root], stream[root]));
        CUDACHECK(hipStreamWaitEvent(stream[left], event[root], 0));

        broadcastMST(d_buffers, count, root, mid, right, stream, event);
        broadcastMST(d_buffers, count, left, left, mid-1, stream, event);
    }
}

/**
 * Notes
 *  - All timers are started in a group before the copies so that any
 *    serialization is caught by the later timers.
 */

using Float = double;

int main(int argc, char *argv[]) {
    const size_t COUNT = (argc > 1) ? atoi(argv[1]) : defaultCount;
    const size_t BUF_SIZE = COUNT * sizeof(Float);
    const int nIters = 10;

    int nDevs;
    CUDACHECK(hipGetDeviceCount(&nDevs));

    std::vector<Float*> d_buffers(nDevs);

    for (int d = 0; d < nDevs; ++d) {
        CUDACHECK(hipSetDevice(d));
        CUDACHECK(hipMalloc(&d_buffers[d], BUF_SIZE));
        for (int p = 0; p < nDevs; ++p) {
            if (p != d) {
                int canAccessPeer;
                CUDACHECK(hipDeviceCanAccessPeer(&canAccessPeer, d, p));
                if (canAccessPeer) {
                    std::cout << "enabling peer access from dev " << d << " to " << p << std::endl;
                    CUDACHECK(hipDeviceEnablePeerAccess(p, 0));
                }
            }
        }
    }

    Float *h_buffer, *h_result;
    CUDACHECK(hipHostMalloc(&h_buffer, BUF_SIZE));
    CUDACHECK(hipHostMalloc(&h_result, BUF_SIZE));
    for (size_t i = 0; i < COUNT; ++i) {
        h_buffer[i] = i;
    }
    CUDACHECK(hipMemcpy(d_buffers[0], h_buffer, BUF_SIZE, hipMemcpyDefault));


    /*
     * Source GPU pushing with stream concurrency
     */

    {
        std::cout << std::endl;
        std::cout << "Source push with N streams" << std::endl;

        std::vector<hipStream_t> streams(nDevs);
        std::vector<hipEvent_t> start(nDevs);
        std::vector<hipEvent_t> stop(nDevs);
        std::vector<float> times(nDevs);

        CUDACHECK(hipSetDevice(0));
        for (int d = 0; d < nDevs; ++d) {
            CUDACHECK(hipStreamCreateWithFlags(&streams[d], hipStreamNonBlocking));
            CUDACHECK(hipEventCreate(&start[d]));
            CUDACHECK(hipEventCreate(&stop[d]));
        }

        for (int iter = 0; iter < nIters; ++iter) {
            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipMemset(d_buffers[d], 0, BUF_SIZE));
            }

            CUDACHECK(hipSetDevice(0));
            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipEventRecord(start[d], streams[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipMemcpyAsync(d_buffers[d], d_buffers[0], BUF_SIZE,
                            hipMemcpyDefault, streams[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipEventRecord(stop[d], streams[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                float ms;
                CUDACHECK(hipStreamSynchronize(streams[d]));
                CUDACHECK(hipEventElapsedTime(&ms, start[d], stop[d]));
                times[d] += ms;
            }
        }

        for (int d = 1; d < nDevs; ++d) {
            printf("dev %d: %f ms\n", d, times[d] / nIters);
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipMemcpy(h_result, d_buffers[d], BUF_SIZE, hipMemcpyDefault));
            checkClose(h_result, h_buffer, COUNT);
        }

        CUDACHECK(hipSetDevice(0));
        for (int d = 0; d < nDevs; ++d) {
            CUDACHECK(hipStreamDestroy(streams[d]));
            CUDACHECK(hipEventDestroy(start[d]));
            CUDACHECK(hipEventDestroy(stop[d]));
        }
    }


    /*
     * Dest GPUs pulling on a single stream per GPU
     */

    {
        std::cout << std::endl << "Destinations pull" << std::endl;

        std::vector<hipEvent_t> start(nDevs);
        std::vector<hipEvent_t> stop(nDevs);
        std::vector<hipStream_t> stream(nDevs);
        std::vector<float> times(nDevs);

        for (int d = 0; d < nDevs; ++d) {
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking));
            CUDACHECK(hipEventCreate(&start[d]));
            CUDACHECK(hipEventCreate(&stop[d]));
        }

        for (int iter = 0; iter < nIters; ++iter) {
            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipMemset(d_buffers[d], 0, BUF_SIZE));
            }

            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(start[d], stream[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipMemcpyAsync(d_buffers[d], d_buffers[0], BUF_SIZE,
                            hipMemcpyDefault, stream[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(stop[d], stream[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                float ms;
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipStreamSynchronize(stream[d]));
                CUDACHECK(hipEventElapsedTime(&ms, start[d], stop[d]));
                times[d] += ms;
            }
        }

        for (int d = 1; d < nDevs; ++d) {
            printf("dev %d: %f ms\n", d, times[d] / nIters);
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipMemcpy(h_result, d_buffers[d], BUF_SIZE, hipMemcpyDefault));
            checkClose(h_result, h_buffer, COUNT);
        }

        for (int d = 0; d < nDevs; ++d) {
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipStreamDestroy(stream[d]));
            CUDACHECK(hipEventDestroy(start[d]));
            CUDACHECK(hipEventDestroy(stop[d]));
        }
    }


    /*
     * Launch a thread per device to do the sends (like NCCL)
     */

    {
        std::cout << std::endl << "Thread per device to pull" << std::endl;

        std::vector<std::thread> threads(nDevs);
        std::vector<float> times(nDevs);
        std::vector<hipStream_t> stream(nDevs);
        std::vector<hipEvent_t> start(nDevs);
        std::vector<hipEvent_t> stop(nDevs);

        for (int d = 0; d < nDevs; ++d) {
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking));
            CUDACHECK(hipEventCreate(&start[d]));
            CUDACHECK(hipEventCreate(&stop[d]));
            times[d] = 0.f;
        }

        for (int i = 0; i < nIters; ++i) {
            for (int d = 0; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(start[d], stream[d]));
            }

            for (int d = 0; d < nDevs; ++d) {
                threads[d] = std::thread(cudaCopy<Float>, d, stream[d], d_buffers[d], d_buffers[0], BUF_SIZE);
            }

            for (auto &thread : threads) {
                thread.join();
            }

            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(stop[d], stream[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                float ms;
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipStreamSynchronize(stream[d]));
                CUDACHECK(hipEventElapsedTime(&ms, start[d], stop[d]));
                times[d] += ms;
            }
        }

        for (int d = 1; d < nDevs; ++d) {
            printf("dev %d: %f ms\n", d, times[d] / nIters);
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipMemcpy(h_result, d_buffers[d], BUF_SIZE, hipMemcpyDefault));
            checkClose(h_result, h_buffer, COUNT);
            CUDACHECK(hipEventDestroy(stop[d]));
            CUDACHECK(hipEventDestroy(start[d]));
            CUDACHECK(hipStreamDestroy(stream[d]));
        }
    }


    /*
     * Dest/source in push/pull (half each)
     */

    const size_t HALF_COUNT = COUNT / 2;
    const size_t HALF_BUF_SIZE = HALF_COUNT * sizeof(Float);

    {
        std::cout << std::endl << "Dest/source push/pull, split half/half" << std::endl;

        std::vector<hipStream_t> stream(nDevs);
        std::vector<hipStream_t> stream0(nDevs);
        std::vector<hipEvent_t> start(nDevs);
        std::vector<hipEvent_t> start0(nDevs);
        std::vector<hipEvent_t> stop(nDevs);
        std::vector<hipEvent_t> stop0(nDevs);
        std::vector<float> times(nDevs);
        std::vector<float> times0(nDevs);

        CUDACHECK(hipSetDevice(0));
        for (int d = 0; d < nDevs; ++d) {
            CUDACHECK(hipStreamCreate(&stream0[d]));
            CUDACHECK(hipEventCreate(&start0[d]));
            CUDACHECK(hipEventCreate(&stop0[d]));
            times0[d] = 0.f;
        }

        for (int d = 1; d < nDevs; ++d) {
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking));
            CUDACHECK(hipEventCreate(&start[d]));
            CUDACHECK(hipEventCreate(&stop[d]));
            times[d] = 0.f;
        }

        for (int i = 0; i < nIters; ++i) {
            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(start[d], stream[d]));
            }

            CUDACHECK(hipSetDevice(0));
            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipEventRecord(start0[d], stream0[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                // push from 0
                CUDACHECK(hipSetDevice(0));
                CUDACHECK(hipMemcpyAsync(d_buffers[d], d_buffers[0],
                            HALF_BUF_SIZE, hipMemcpyDefault, stream0[d]));

                // pull from dev d
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipMemcpyAsync(d_buffers[d]+HALF_COUNT, d_buffers[0]+HALF_COUNT,
                            (BUF_SIZE-HALF_BUF_SIZE), hipMemcpyDefault, stream[d]));
            }

            CUDACHECK(hipSetDevice(0));
            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipEventRecord(stop0[d], stream0[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(stop[d], stream[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                float ms;
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipStreamSynchronize(stream[d]));
                CUDACHECK(hipEventElapsedTime(&ms, start[d], stop[d]));
                times[d] += ms;
            }

            CUDACHECK(hipSetDevice(0));
            for (int d = 1; d < nDevs; ++d) {
                float ms;
                CUDACHECK(hipStreamSynchronize(stream0[d]));
                CUDACHECK(hipEventElapsedTime(&ms, start0[d], stop0[d]));
                times0[d] += ms;
            }
        }

        for (int d = 1; d < nDevs; ++d) {
            printf("dev 0 to dev %d: %f ms\n", d, times0[d] / nIters);
        }

        for (int d = 1; d < nDevs; ++d) {
            printf("dev %d: %f ms\n", d, times[d] / nIters);
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipMemcpy(h_result, d_buffers[d], BUF_SIZE, hipMemcpyDefault));
            checkClose(h_result, h_buffer, COUNT);
            CUDACHECK(hipEventDestroy(stop[d]));
            CUDACHECK(hipEventDestroy(start[d]));
            CUDACHECK(hipStreamDestroy(stream[d]));
        }
    }


    /*
     * Dest/source in push/pull (half each), with separate threads per device
     */

    {
        std::cout << std::endl
            << "Dest/source push/pull, separate threads per device" << std::endl;

        std::vector<hipStream_t> stream(nDevs);
        std::vector<hipStream_t> stream0(nDevs);
        std::vector<hipEvent_t> start(nDevs);
        std::vector<hipEvent_t> start0(nDevs);
        std::vector<hipEvent_t> stop(nDevs);
        std::vector<hipEvent_t> stop0(nDevs);
        std::vector<float> times(nDevs);
        std::vector<float> times0(nDevs);
        std::vector<std::thread> threads(nDevs);

        CUDACHECK(hipSetDevice(0));
        for (int d = 0; d < nDevs; ++d) {
            CUDACHECK(hipStreamCreate(&stream0[d]));
            CUDACHECK(hipEventCreate(&start0[d]));
            CUDACHECK(hipEventCreate(&stop0[d]));
            times0[d] = 0.f;
        }

        for (int d = 1; d < nDevs; ++d) {
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking));
            CUDACHECK(hipEventCreate(&start[d]));
            CUDACHECK(hipEventCreate(&stop[d]));
            times[d] = 0.f;
        }

        for (int i = 0; i < nIters; ++i) {
            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(start[d], stream[d]));
            }

            CUDACHECK(hipSetDevice(0));
            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipEventRecord(start0[d], stream0[d]));
            }

            std::thread root(cudaCopyRoot<Float>, 0, nDevs, &stream0[0],
                    &d_buffers[0], d_buffers[0], HALF_BUF_SIZE);

            for (int d = 1; d < nDevs; ++d) {
                threads[d] = std::thread(cudaCopy<Float>, d, stream[d],
                        d_buffers[d]+HALF_COUNT, d_buffers[0]+HALF_COUNT,
                        (BUF_SIZE-HALF_BUF_SIZE));
            }

            root.join();
            for (int d = 1; d < nDevs; ++d) {
                threads[d].join();
            }

            CUDACHECK(hipSetDevice(0));
            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipEventRecord(stop0[d], stream0[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(stop[d], stream[d]));
            }

            for (int d = 1; d < nDevs; ++d) {
                float ms;
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipStreamSynchronize(stream[d]));
                CUDACHECK(hipEventElapsedTime(&ms, start[d], stop[d]));
                times[d] += ms;
            }

            CUDACHECK(hipSetDevice(0));
            for (int d = 1; d < nDevs; ++d) {
                float ms;
                CUDACHECK(hipStreamSynchronize(stream0[d]));
                CUDACHECK(hipEventElapsedTime(&ms, start0[d], stop0[d]));
                times0[d] += ms;
            }
        }

        CUDACHECK(hipSetDevice(0));
        for (int d = 1; d < nDevs; ++d) {
            printf("dev 0 to dev %d: %f ms\n", d, times0[d] / nIters);
            CUDACHECK(hipEventDestroy(start0[d]));
            CUDACHECK(hipEventDestroy(stop0[d]));
            CUDACHECK(hipStreamDestroy(stream0[d]));
        }

        for (int d = 1; d < nDevs; ++d) {
            printf("dev %d: %f ms\n", d, times[d] / nIters);
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipMemcpy(h_result, d_buffers[d], BUF_SIZE, hipMemcpyDefault));
            checkClose(h_result, h_buffer, COUNT);
            CUDACHECK(hipEventDestroy(stop[d]));
            CUDACHECK(hipEventDestroy(start[d]));
            CUDACHECK(hipStreamDestroy(stream[d]));
        }
    }


    /*
     * Send via MST
     */

    {
        std::cout << std::endl << "MST broadcast" << std::endl;

        std::vector<hipStream_t> stream(nDevs);
        std::vector<hipEvent_t> event(nDevs);
        std::vector<hipEvent_t> start(nDevs);
        std::vector<hipEvent_t> stop(nDevs);
        std::vector<float> times(nDevs);

        for (int d = 0; d < nDevs; ++d) {
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking));
            CUDACHECK(hipEventCreate(&event[d]));
            CUDACHECK(hipEventCreate(&start[d]));
            CUDACHECK(hipEventCreate(&stop[d]));
            times[d] = 0;
        }

        for (int iter = 0; iter < nIters; ++iter) {
            for (int d = 0; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(start[d], stream[d]));
            }

            broadcastMST(&d_buffers[0], COUNT, 0, 0, nDevs-1, &stream[0], &event[0]);

            for (int d = 0; d < nDevs; ++d) {
                CUDACHECK(hipSetDevice(d));
                CUDACHECK(hipEventRecord(stop[d], stream[d]));
                CUDACHECK(hipStreamSynchronize(stream[d]));
                float ms;
                CUDACHECK(hipEventElapsedTime(&ms, start[d], stop[d]));
                times[d] += ms;
            }
        }

        for (int d = 0; d < nDevs; ++d) {
            printf("dev %d: %f ms\n", d, times[d] / nIters);
            CUDACHECK(hipSetDevice(d));
            CUDACHECK(hipMemcpy(h_result, d_buffers[d], BUF_SIZE, hipMemcpyDefault));
            checkClose(h_result, h_buffer, COUNT);
            CUDACHECK(hipEventDestroy(stop[d]));
            CUDACHECK(hipEventDestroy(start[d]));
            CUDACHECK(hipEventDestroy(event[d]));
            CUDACHECK(hipStreamDestroy(stream[d]));
        }
    }


    /*
     * Clean up
     */

    for (int d = 0; d < nDevs; ++d) {
        CUDACHECK(hipFree(d_buffers[d]));
    }
    CUDACHECK(hipHostFree(h_buffer));
    CUDACHECK(hipHostFree(h_result));

    return 0;
}
