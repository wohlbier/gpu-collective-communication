#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <string>
#include <vector>

#include "common.h"

static const size_t COUNT = 10000000;

std::string horLine = "------------------------------------------------------------\n";

void reportBandwidth(std::string testStr, float *ms, const int N) {
    // pull out the max time - bandwidth determined by slowest copy
    float max = *(std::max_element(ms, ms + N));
    size_t bytes = COUNT * sizeof(tx_type);
    float bw = (bytes/(1024*1024)) / max;

    std::cout << horLine << testStr << std::endl;
    std::cout << "\tbw: " << bw << " GBps" << std::endl;
    std::cout << horLine << std::endl;
}

void exitGpuCount(const int need, const int have) {
    std::cout << "not enough GPUs to continue" << std::endl;
    std::cout << "\tneed: " << need << ", have: " << have << std::endl;
    exit(EXIT_FAILURE);
}

int main(int argc, char *argv[]) {
    // get how many devices I have to work with
    int nDevs;
    CUDACHECK(hipGetDeviceCount(&nDevs));

    // need at least 2 GPUs from here on out
    if (nDevs < 2) {
        exitGpuCount(2, nDevs);
    }

    /*
     * Copy 0 -> 1
     *  - First do solely 0 -> 1
     *  - Then split evenly so that 0 pushes and 1 pulls
     */

    std::vector<tx_type*> d_buffers(nDevs);
    std::vector<hipStream_t> stream0(nDevs);
    std::vector<hipStream_t> stream1(nDevs);
    std::vector<hipEvent_t> start(nDevs);
    std::vector<hipEvent_t> stop(nDevs);

    for (int i = 0; i < nDevs; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipStreamCreateWithFlags(&stream0[i], hipStreamNonBlocking));
        CUDACHECK(hipStreamCreateWithFlags(&stream1[i], hipStreamNonBlocking));
        CUDACHECK(hipEventCreate(&start[i]));
        CUDACHECK(hipEventCreate(&stop[i]));
    }

    const size_t numBytes = COUNT * sizeof(tx_type);
    tx_type *h_data, *h_result;
    CUDACHECK(hipHostMalloc(&h_data, numBytes));
    CUDACHECK(hipHostMalloc(&h_result, numBytes));
    CUDACHECK(hipMemset(h_data, 1., numBytes));
    CUDACHECK(hipMemset(h_result, 0., numBytes));

    for (int i = 0; i < 2; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipMalloc(&d_buffers[i], numBytes));
    }
    CUDACHECK(hipMemcpy(d_buffers[0], h_data, numBytes, hipMemcpyDefault));

    // copy
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipEventRecord(start[0]));
    CUDACHECK(hipMemcpyAsync(d_buffers[1], d_buffers[0], numBytes, hipMemcpyDefault));
    CUDACHECK(hipEventRecord(stop[0]));

    // get timing
    float ms;
    CUDACHECK(hipEventSynchronize(stop[0]));
    CUDACHECK(hipEventElapsedTime(&ms, start[0], stop[0]));
    reportBandwidth("0 push all to 1", &ms, 1);

    // make sure the data got there properly
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMemcpy(h_result, d_buffers[1], numBytes, hipMemcpyDefault));
    for (size_t i = 0; i < COUNT; ++i) {
        if (h_result[i] != h_data[i]) {
            fprintf(stderr, "data mismatch: orig[%ld]=%f, copy[%ld]=%f\n",
                    i, h_data[i], i, h_result[i]);
            exit(EXIT_FAILURE);
        }
    }

    // Even split between 0 push to 1 and 1 pull from 0

    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMemset(d_buffers[1], 0, numBytes));

    const size_t half = COUNT / 2;
    const size_t halfBytes = half * sizeof(tx_type);
    float twoMs[2] = { 0 };

    //// start the timers
    //for (int i = 0; i < 2; ++i) {
    //    CUDACHECK(hipSetDevice(i));
    //    CUDACHECK(hipEventRecord(start[i]));
    //}

    //// push first half with dev 0
    //CUDACHECK(hipSetDevice(0));
    //CUDACHECK(hipMemcpyAsync(d_buffers[1], d_buffers[0], halfBytes, hipMemcpyDefault));

    //// pull second half with dev 1
    //CUDACHECK(hipSetDevice(1));
    //CUDACHECK(hipMemcpyAsync(d_buffers[1] + half, d_buffers[0] + half,
    //            numBytes - halfBytes, hipMemcpyDefault));

    //// stop timers
    //for (int i = 0; i < 2; ++i) {
    //    CUDACHECK(hipSetDevice(i));
    //    CUDACHECK(hipEventRecord(stop[i]));
    //}

    //// get timing
    //for (int i = 0; i < 2; ++i) {
    //    CUDACHECK(hipSetDevice(i));
    //    CUDACHECK(hipEventSynchronize(stop[i]));
    //    CUDACHECK(hipEventElapsedTime(&twoMs[i], start[i], stop[i]));
    //}
    //reportBandwidth("0 push half, 1 pull half", twoMs, 2);

    CUDACHECK(hipSetDevice(0));

    // create some new events for this particular one
    std::vector<hipEvent_t> twoStart(2);
    std::vector<hipEvent_t> twoStop(2);

    for (int i = 0; i < 2; ++i) {
        CUDACHECK(hipEventCreate(&twoStart[i]));
        CUDACHECK(hipEventCreate(&twoStop[i]));
    }

    CUDACHECK(hipEventRecord(twoStart[0], stream0[0]));
    CUDACHECK(hipEventRecord(twoStart[1], stream1[0]));

    CUDACHECK(hipMemcpyAsync(d_buffers[1], d_buffers[0], halfBytes,
                hipMemcpyDefault, stream0[0]));
    CUDACHECK(hipMemcpyAsync(d_buffers[1]+half, d_buffers[0]+half, halfBytes,
                hipMemcpyDefault, stream1[0]));

    CUDACHECK(hipEventRecord(twoStop[0], stream0[0]));
    CUDACHECK(hipEventRecord(twoStop[1], stream1[0]));

    CUDACHECK(hipStreamSynchronize(stream0[0]));
    CUDACHECK(hipStreamSynchronize(stream1[0]));

    CUDACHECK(hipEventElapsedTime(&twoMs[0], twoStart[0], twoStop[0]));
    CUDACHECK(hipEventElapsedTime(&twoMs[1], twoStart[1], twoStop[1]));

    std::cout << twoMs[0] << "  " << twoMs[1] << std::endl;
    reportBandwidth("half and half", twoMs, 2);

    // make sure the data got there properly
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMemcpy(h_result, d_buffers[1], numBytes, hipMemcpyDefault));
    for (size_t i = 0; i < COUNT; ++i) {
        if (h_result[i] != h_data[i]) {
            fprintf(stderr, "data mismatch: orig[%ld]=%f, copy[%ld]=%f\n",
                    i, h_data[i], i, h_result[i]);
            exit(EXIT_FAILURE);
        }
    }

    /*
     * cleanup
     */

    // cuda things
    CUDACHECK(hipHostFree(h_data));
    CUDACHECK(hipHostFree(h_result));
    for (int i = 0; i < 2; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipFree(d_buffers[i]));
    }

    return 0;
}
